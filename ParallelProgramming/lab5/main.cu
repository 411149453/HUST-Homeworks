#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <cstdlib>
#include <cstdio>
#include "../lib/CImg-2.3.3/CImg.h"

#define min(a, b) ((a) < (b) ? (a) : (b))
#define max(a, b) ((a) > (b) ? (a) : (b))

using namespace std;
using namespace cimg_library;

#define maskWidth 5
#define maskHeight 5


struct Work {
    int rowStart;
    int rowEnd;
    int colStart;
    int colEnd;
};

const unsigned char erodeMask[maskWidth * maskHeight] = {
    0, 0, 1, 0, 0,
    0, 0, 1, 0, 0,
    1, 1, 1, 1, 1,
    0, 0, 1, 0, 0,
    0, 0, 1, 0, 0,
};
const unsigned char dilateMask[maskWidth * maskHeight] = {
    1, 1, 1, 1, 1,
    1, 1, 1, 1, 1,
    1, 1, 1, 1, 1,
    1, 1, 1, 1, 1,
    1, 1, 1, 1, 1,
};

//const unsigned char erodeMask[maskWidth * maskHeight] = {
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//}, dilateMask[maskWidth * maskHeight] = {
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//    0, 0, 0, 0, 0,
//};

__global__ void pixelProcessErodeKernel(
        int blockWidth,
        int blockHeight,
        int binaryImageWidth,
        int binaryImageHeight,
        int expandedImageWidth,
        unsigned char *expandedImage,
        unsigned char *erodedImage,
        unsigned char *erodeMask
        ){
    // get cuda parameter
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int x = threadIdx.x;
    int y = threadIdx.y;

    // calculate index
    int blocksPerRow = (binaryImageWidth / blockWidth + 1);
    int blocksPerCol = (binaryImageHeight/ blockHeight + 1);
    int col = (blockX % blocksPerRow) * blockWidth + x;
    int row = (blockY % blocksPerCol) * blockHeight + y;
    int index = row * binaryImageWidth + col;

    unsigned char erodeResult = 1;
    unsigned char dilateResult = 1;

    if(col < binaryImageWidth && row < binaryImageHeight){
        for(int maskRow=0; maskRow < maskHeight; ++maskRow){
            for(int maskCol=0; maskCol < maskWidth; ++maskCol){
                int expandedIndex = (row + maskRow) * expandedImageWidth + (col + maskCol);
                erodeResult &= (expandedImage[expandedIndex] |
                        erodeMask[maskRow * maskWidth + maskCol]);
            }
        }
    }

    erodedImage[index] = erodeResult;
}

__global__ void pixelProcessDilateKernel(
        int blockWidth,
        int blockHeight,
        int binaryImageWidth,
        int binaryImageHeight,
        int expandedImageWidth,
        unsigned char *expandedImage,
        unsigned char *dilatedImage,
        unsigned char *dilateMask
        ){
    // get cuda parameter
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int x = threadIdx.x;
    int y = threadIdx.y;

    // calculate index
    int blocksPerRow = (binaryImageWidth / blockWidth + 1);
    int blocksPerCol = (binaryImageHeight/ blockHeight + 1);
    int col = (blockX % blocksPerRow) * blockWidth + x;
    int row = (blockY % blocksPerCol) * blockHeight + y;
    int index = row * binaryImageWidth + col;

    unsigned char dilateResult = 0;

    if(col < binaryImageWidth && row < binaryImageHeight){
        for(int maskRow=0; maskRow < maskHeight; ++maskRow){
            for(int maskCol=0; maskCol < maskWidth; ++maskCol){
                int expandedIndex = (row + maskRow) * expandedImageWidth + (col + maskCol);
                dilateResult |= (expandedImage[expandedIndex] &
                        dilateMask[maskRow * maskWidth + maskCol]);
            }
        }
    }

    dilatedImage[index] = dilateResult;
}

int main(int argc, char *argv[]) {
    int blockSize = 32;
    for (int i = 1; i < argc; ++i) {
        if (!strcmp(argv[i], "-s"))
            blockSize = atoi(argv[++i]);
    }
    cout << "block size " << blockSize << endl;

    CImg<unsigned char> srcImage("../data/Lenna.png");

    // Convert RGB image to grayscale image
    CImg<unsigned char> grayScaleImage(
            srcImage.width(), srcImage.height(), 1, 1);
    unsigned char r, g, b;
    unsigned char gr1 = 0;
    for (int i = 0; i < srcImage.width(); ++i) {
        for (int j = 0; j < srcImage.height(); ++j) {
            //Return a pointer to a located pixel value.
            r = srcImage(i, j, 0, 0); // First channel RED
            g = srcImage(i, j, 0, 1); // Second channel GREEN
            b = srcImage(i, j, 0, 2); // Third channel BLUE

            //PAL and NTSC
            //Y = 0.299*R + 0.587*G + 0.114*B
            gr1 = round(0.299 * ((double)r) + 0.587 * ((double)g) + 0.114 * ((double)b));

            // assign the luminosity value
            grayScaleImage(i, j, 0, 0) = gr1;
        }
    }

    // binarization
    CImg<unsigned char> binaryImage = grayScaleImage.get_threshold(128);

    // expansion
    CImg<unsigned char> expandedImage = binaryImage.get_resize(
            binaryImage.width() + maskWidth - 1,
            binaryImage.height() + maskHeight - 1,
            1, 1,
            0, 0, // fill border with black
            0.5, 0.5);

    // do erode and dilate
    CImg<unsigned char>
        erodedImage(binaryImage.width(), binaryImage.height(), 1, 1),
        dilatedImage(binaryImage.width(), binaryImage.height(), 1, 1);

    //    // split work
    //    Work works[(binaryImage.height() / blockSize + 1) *
    //        (binaryImage.width() / blockSize + 1)];
    //    int workNum = 0;
    //    for (int row = 0; row < binaryImage.height(); row += blockSize) {
    //        for(int col=0; col < binaryImage.width(); col += blockSize){
    //            works[workNum].colStart = col;
    //            works[workNum].colEnd = min(col + blockSize, binaryImage.width());
    //            works[workNum].rowStart = row;
    //            works[workNum].rowEnd = min(row + blockSize, binaryImage.height());
    //            ++workNum;
    //        }
    //    }

    unsigned char *cudaErodeMask, *cudaDilateMask;
    int maskSize = sizeof(unsigned char) * maskWidth * maskHeight;
    hipMalloc(&cudaErodeMask, maskSize);
    hipMalloc(&cudaDilateMask, maskSize);
    hipMemcpy(cudaErodeMask, erodeMask, maskSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaDilateMask, dilateMask, maskSize, hipMemcpyHostToDevice);

    for (int repeater = 0; repeater < 1000; ++repeater) {
        int result;

        // malloc graphics ram for image, move the image to cuda
        unsigned char *cudaExpandedImage, *cudaErodedImage, *cudaDilatedImage;
        int expandedImageSize = sizeof(unsigned char) * expandedImage.size();
        int binaryImageSize = sizeof(unsigned char) * binaryImage.size();
        hipMalloc(&cudaExpandedImage, expandedImageSize);
        hipMemcpy(cudaExpandedImage, expandedImage.data(),
                expandedImageSize, hipMemcpyHostToDevice);

        hipMalloc(&cudaErodedImage, binaryImageSize);
        hipMalloc(&cudaDilatedImage, binaryImageSize);

        dim3 thread(blockSize, blockSize);
        dim3 block(binaryImage.width() / blockSize + 1,
                binaryImage.height() / blockSize + 1);

        pixelProcessErodeKernel<<<block, thread>>>(
                blockSize, blockSize,
                binaryImage.width(), binaryImage.height(),
                expandedImage.width(), cudaExpandedImage,
                cudaErodedImage, cudaErodeMask);

        pixelProcessDilateKernel<<<block, thread>>>(
                blockSize, blockSize,
                binaryImage.width(), binaryImage.height(),
                expandedImage.width(), cudaExpandedImage,
                cudaDilatedImage, cudaDilateMask);

        hipMemcpy(erodedImage.data(), cudaErodedImage,
                binaryImageSize, hipMemcpyDeviceToHost);
        hipMemcpy(dilatedImage.data(), cudaDilatedImage,
                binaryImageSize, hipMemcpyDeviceToHost);

        hipFree(cudaExpandedImage);
        hipFree(cudaErodedImage);
        hipFree(cudaDilatedImage);

        cout << "round " << repeater << " finished" << endl;
    }

    hipFree(cudaErodeMask);
    hipFree(cudaDilateMask);

    binaryImage.normalize(0, 255);
    binaryImage.save("binary.png");

    erodedImage.normalize(0, 255);
    erodedImage.save("eroded.png");

    dilatedImage.normalize(0, 255);
    dilatedImage.save("dilated.png");

    return 0;
}

